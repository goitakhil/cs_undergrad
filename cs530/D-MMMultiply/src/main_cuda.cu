#include "hip/hip_runtime.h"
#include "matrix_matrix.hpp"
#include "Configuration.hpp"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// TODO: Add cmdline arg checks for file names
// TODO: Add command arg/flag for thread limit

void freeMatrix(CudaMatrix* matrix);

int main(int argc, char** argv){
  //fprintf(stderr,"Running matrix_matrix main\n");
  int debugMode = 0;

  // Configuration and cmdline parser is from office hours meeting
  /*Configuration config;
  config.addParamString("filename1",'a',NULL,"-a [string] the first matrix");
  config.addParamString("filename2",'b',NULL,"-b [string] the second matrix");

  config.parse(argc,argv);

  printf("I am planning to read: %s and %s\n",
         config.getString("filename1").c_str(),
         config.getString("filename2").c_str());*/
	
  if (argc == 4 && strcmp(argv[3], "-d") == 0) {
    // Turn on debug mode
    printf("Turning on debug mode with %s\n", argv[3]);
    debugMode = 1;
  }
  else if(argc != 3){
    fprintf(stderr,"ERROR: You had %d command arguments.\n", argc);
    fprintf(stderr,"USAGE: ./matrix_matrix <matrixA.txt> <matrixB.txt>\n");
    return 0;
  }

  
  struct CudaMatrix* a = read_cuda_matrix_file(argv[1]);
  struct CudaMatrix* b = read_cuda_matrix_file(argv[2]);

  // Cuda, grid dimension from Nvidia docs. Link in README.
  dim3 dimBlock(a->rows/2, b->cols/2); // Using max/2 so that it's not overkill 
  dim3 dimGrid(b->cols / dimBlock.x, a->rows / dimBlock.y); // From Nvidia docs
  
  // Cuda
  //int blockSize = 2; // TODO: What size to use?
  //int numBlocks = (a->rows*b->cols + blockSize - 1) / blockSize; // TODO: Determine if this is the right calculation to use

  //printf("numBlocks: %d\n", numBlocks);

  // Allocing the c matrix with cuda
  struct CudaMatrix *c;
  hipMallocManaged(&c, sizeof(struct Matrix));
  hipMallocManaged(&c->data, a->rows*b->cols*sizeof(double));
  c->rows = a->rows;
  c->cols = b->cols;

  //matrix_matrix_mult<<<numBlocks, blockSize>>>(a, b, c, debugMode);
  matrix_matrix_mult<<<dimGrid, dimBlock>>>(a, b, c, debugMode); // Using 1,1 until cuda works
  
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  if (c == NULL) return 0;
  //printf("Got back to main\n");
  //printf("[%lf, %lf, %lf, %lf]\n", c->data[0][0], c->data[0][1], c->data[1][0], c->data[1][1]);
  
  write_cuda_matrix_file(c);

  freeMatrix(a);
  freeMatrix(b);
  freeMatrix(c);
  
  return 0;
}

void freeMatrix(CudaMatrix* matrix) {
  
  //free(matrix->data);
  //free(matrix);
  hipFree(matrix->data);
  hipFree(matrix);
}
