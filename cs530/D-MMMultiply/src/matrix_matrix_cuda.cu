#include "hip/hip_runtime.h"
#include "matrix_matrix.hpp"
#include "timing.c"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdexcept>
#include <errno.h>

#define MAX_LENGTH 2048
#define MAX_TOKENS 1000

int parseCommandLine(int argc, char** argv){
  return 0;
}

// Parsing code is from CS253-resources
char **parseInput(char *s, char *delimiter, int *numTokens);

__global__
void matrix_matrix_mult(struct CudaMatrix* A, struct CudaMatrix* B, struct CudaMatrix* C, int debugMode){
/*  if (A == NULL || B == NULL) {
    throw std::invalid_argument("Can't multiply a NULL matrix.");
  }

  if (A->cols != B->rows) {
    throw std::invalid_argument("A's rows and B's columns don't match.");
  }
*/

  // Cuda offsets 
  /* 
  int iOffset = blockDim.x * blockIdx.x;
  int jOffset = blockDim.y * blockIdx.y;

  int i;
  for (i=iOffset; i<gridDim.x*blockDim.x; i+=iOffset) { // block/grid x dim
    int j;
    for (j=jOffset; j<gridDim.y*blockDim.y; j+=jOffset) { // block/grid y dim
      int k;
      for (k=0; k<A->cols; k++) { // cols in a
        double a = A->data[i][k];
        double b = B->data[k][j];
        C->data[i][j] += a * b;
      }
    }
  }

*/
  //int index = threadIdx.x; //index of the current thread within it's block
  //int stride = blockDim.x; //number of threads in the block.

//printf("threadIdx: %d\n", index);
//printf("stride: %d\n", stride);
 
  /*
  int i;
  for (i = index; i < A->rows; i += stride) {
    int j;
    for(j = 0; j < B->cols; j++) {
      int k;
      for(k = 0; k < A->cols; k++) {
        C->data[i * B->cols + j] += A->data[i * A->cols + k] * B->data[k * B->cols + j]; 
      }
    }  
  }*/

  // This code is from a Nvidia developer documentation. Link in README.
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int k;
  for(k = 0; k < A->cols; k++) {
    C->data[row * B->cols + col] += A->data[row * A->cols + k] * B->data[k * B->cols + col]; 
  }



  /*
  // Original serial code
  int i;
  start_time = getMilliSeconds();
  for (i=0; i<A->rows; i++) { // rows in a
    C->data[i] = (double*)malloc(B->cols*sizeof(double)); // malloc'ing the C matrix columns
    int j;
    for (j=0; j<B->cols; j++) { // cols in b
      C->data[i][j] = 0;
      int k;
      for (k=0; k<A->cols; k++) { // cols in a
        double a = A->data[i][k];
        double b = B->data[k][j];
        C->data[i][j] += a * b;
      }
    }
  }*/

  //Had to get rid to timing stuff because cuda doesn't like it. 
}

struct CudaMatrix* read_cuda_matrix_file(char* filename){
  printf("Read matrix file: %s \n", filename);
  
  FILE *file = fopen(filename, "r");
  if(!file){
    fprintf(stderr, "An error has occurred. Wrong file name?\n");
    exit(errno);
  }

  // Parse - Code is from CS253-resources and C-MVMultiply.
  double* data = (double*)malloc(sizeof(double)*MAX_TOKENS); 
  int numTokens = 0;
  char *delim = strdup(" ");
  char buffer[MAX_LENGTH];
  char **token;
  int loop = 0;
  while(fgets(buffer, sizeof(buffer), file)!=NULL){
    token = parseInput(buffer, delim, &numTokens);
    int i;
    for(i=0; i<numTokens; i++){
      data[i+loop]=strtod(token[i], NULL);
   	}
    loop+=numTokens;       
  }
    
  // Build matrix with data
  int rows = (int) data[0];
  int cols = (int) data[1];

  // Mallocing matrix with cuda 
  struct CudaMatrix *matrix;
  hipMallocManaged(&matrix, sizeof(struct Matrix));
  hipMallocManaged(&matrix->data, rows*cols*sizeof(double));
  matrix->rows = rows;
  matrix->cols = cols;
  
  int b;
  int dataIndex = 2; // 2 to account for index 0 and 1 are taken for row and col values.
  for (b=0; b<rows*cols; b++){
      matrix->data[b] = data[dataIndex++];
  }
  
  fclose(file); // This line in mentioned in the gdb stacktrace?
  return matrix;
}


// The parseInput() function is from CS253-resources.
char **parseInput(char *s, char *delimiter, int *numTokens) {
  char *nextToken;
  char **token = (char **) malloc (sizeof(char *) * MAX_TOKENS);

  // Tokenize the string s 
  nextToken = strtok(s, delimiter);
  *numTokens = 0;
  while (nextToken != NULL) { // && *numTokens < MAX_TOKENS
    token[*numTokens] = (char *) malloc(sizeof(char) * (strlen(nextToken)+1));
    strcpy(token[*numTokens], nextToken);
    (*numTokens)++;
    nextToken = strtok(NULL, delimiter);
  }
  // Now the tokens are copied into token[0..numTokens-1]; 
  return token;
}


void write_cuda_matrix_file(struct CudaMatrix* matrix){
  FILE *fout = fopen("result_cuda.txt", "w");
  int i,j;
  
  // Print matrix dimensions
  fprintf(fout, "%d\n", matrix->rows);
  fprintf(fout, "%d\n", matrix->cols);
  
  // Print matrix data
  for (i=0; i< matrix->rows; i++){
    for (j=0; j<matrix->cols; j++) {
      fprintf(fout, "%lf", matrix->data[i * matrix->cols + j]);
      if (j != matrix->cols-1) fprintf(fout, " "); // Adds spaces between data values
    }
    fprintf(fout, "\n");
  }
  fclose(fout);  
}
